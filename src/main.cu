#include <iostream>
#include <chrono>
#include "vectorCPU.h"
#include "vectorGPU.h"


int N = 10000000;


// Host main routine0
int main(int argc, char** argv){
if(argc != 2){
		std::cout << "Type number of elements ... using default N = " << N << std::endl;
} else {
N = atoi(argv[1]);
}

int nDevices;
hipGetDeviceCount(&nDevices);

if (nDevices == 0){
	std::cout << "No CUDA Devices found. Exiting" << std::endl;
	exit(-1);
}
for (int i = 0;i<nDevices;++i){
hipDeviceProp_t prop;
hipGetDeviceProperties(&prop,i);
std::cout << "Device Number: " << i << std::endl;
std::cout << "Memory Bus Width: " << prop.memoryBusWidth << std::endl;
std::cout << "Memory Clock Rate: " << prop.memoryClockRate << std::endl;
std::cout << "Compute Mode: " << prop.computeMode << std::endl;
std::cout << "Clock Rate: " << prop.clockRate << std::endl;
std::cout << "Total Global Memory: " << prop.totalGlobalMem / 1048576.0f << " MBytes" << std::endl;
std::cout << "Total Shared Memory / Block : " << prop.sharedMemPerBlock << " Bytes" << std::endl;
std::cout << "Warp Size: " << prop.warpSize << std::endl;
std::cout << "MultiProcessor Count: " << prop.multiProcessorCount << std::endl;
std::cout << "Max Threads per multiprocessor: " << prop.maxThreadsPerMultiProcessor << std::endl;
std::cout << "Max Threads per block: " << prop.maxThreadsPerBlock << std::endl;
std::cout << "Max Dimension of thread block(x,y,z): (" << prop.maxThreadsDim[0] << "," << prop.maxThreadsDim[1] << "," << prop.maxThreadsDim[2] << ")" << std::endl;
std::cout << "Max Dimension of grid size(x,y,z): (" << prop.maxGridSize[0] << "," << prop.maxGridSize[1] << "," << prop.maxGridSize[2] << ")" << std::endl;


}
int driverVersion;
hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&driverVersion);
std::cout << "---------------------" << std::endl;

float *a,*b,*c;
a = (float*)malloc(sizeof(float)*N);
b = (float*)malloc(sizeof(float)*N);
c = (float*)malloc(sizeof(float)*N);

for(int i=0;i<N;++i){
		a[i] = i; b[i] = N-i;

}

hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start, 0);
vectorAddCPU(a,b,c,N);
hipEventRecord(stop, 0);
hipEventSynchronize(stop);
float cpu_elapsed_time_ms;
hipEventElapsedTime(&cpu_elapsed_time_ms, start, stop);

std::cout << "CPU OK -> " << cpu_elapsed_time_ms << " [ms]\n";





// Precisamos alocar memória na GPU

float *d_a,*d_b,*d_c;
hipError_t err;
err = hipMalloc((void**)&d_a,sizeof(float)*N);
if(err != hipSuccess){
	std::cout << "GPU allocation error " << std::endl;
		exit(EXIT_FAILURE);

}

err = hipMalloc((void**)&d_b,sizeof(float)*N);
if(err != hipSuccess){
	std::cout << "GPU allocation error " << std::endl;
		exit(EXIT_FAILURE);

}
err = hipMalloc((void**)&d_c,sizeof(float)*N);
if(err != hipSuccess){
	std::cout << "GPU allocation error " << std::endl;
		exit(EXIT_FAILURE);

}
err = hipMemcpy(d_a,a,sizeof(float)*N,hipMemcpyHostToDevice);
if(err != hipSuccess){
	std::cout << "GPU memCpy error " << std::endl;
		exit(EXIT_FAILURE);

}
err = hipMemcpy(d_b,b,sizeof(float)*N,hipMemcpyHostToDevice);
if(err != hipSuccess){
	std::cout << "GPU memCpy error " << std::endl;
		exit(EXIT_FAILURE);

}



int block_size = 32; // 256
int grid_size = (N + block_size - 1 )/block_size;
std::cout << "grid size = " << grid_size << std::endl;
hipEventRecord(start, 0);
vectorAddGPU<<<grid_size,block_size>>>(d_a,d_b,d_c,N);

err = hipGetLastError();

if(err != hipSuccess){
		std::cout << "CUDA kernel launch error" << std::endl;
		fprintf(stderr,"Error code %s",hipGetErrorString(err));
		exit(EXIT_FAILURE);
}



float * c_fromgpu = (float*)malloc(sizeof(float)*N);

hipEventRecord(stop, 0);
hipMemcpy(c_fromgpu,d_c,sizeof(float)*N,hipMemcpyDeviceToHost);

hipDeviceSynchronize();

// Event

hipEventSynchronize(stop);

float gpu_elapsed_time_ms;
hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);

std::cout << "GPU OK -> " << gpu_elapsed_time_ms << " [ms]\n";
std::cout << "Checking results .. " << std::endl;
 for(int i=0;i<N;++i){
 		if (c[i] != c_fromgpu[i])
 		{
 				std::cout << c[i] << "," << c_fromgpu[i] << std::endl;
 				std::cout << "Wrong results! :( " << i << std::endl;
				exit(EXIT_FAILURE);
 		}
 }

 std::cout << "OK!\n";

// std::cout << "All good!" << std::endl;
// std::cout << "Shared Memory Test" << std::endl;
// float *s_a,*s_b,*s_c;
// hipMallocManaged(&s_a,sizeof(float)*N);
// hipMallocManaged(&s_b,sizeof(float)*N);
// hipMallocManaged(&s_c,sizeof(float)*N);
// // hipProfilerStop();
// // return 0; // Profiling won't work for shared memory
// for(int i=0;i<N;++i){
// s_a[i] = 1; s_b[i] = 2;
// }
// vectorAddGPU<<<grid_size,block_size>>>(s_a,s_b,s_c,N);
// hipDeviceSynchronize();

//  for(int i=0;i<N;++i){
//  		if (c[i] != s_c[i]){
//  				std::cout << c[i] << "," << s_c[i] << std::endl; // We can access as CPU memory.
//  				std::cout << "Wrong Results .. :(" << i << std::endl;
// 		exit(EXIT_FAILURE);
//  		}
//  }

// std::cout << "All good shared! " << std::endl;
std::cout << "Total memory used -> " << 3*sizeof(float)*N / (1000*1000) << " MB." << std::endl;
// std::cout << "Total GPU memory used -> " << 6*sizeof(float)*N / (1000*1000) << " MB." << std::endl;
std::cout << "Press enter to exit!" << std::endl;
std::cin.get();

free(a);
free(b);
free(c);
free(c_fromgpu);

hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
// hipFree(s_a);
// hipFree(s_b);
// hipFree(s_c);
return 0; 

}
