#include <iostream>
#include <chrono>

#include "vectorCPU.h"
#include "vectorGPU.h"
#define N 100000000 
// Host main routine0
int main(){

float *a,*b,*c;
a = (float*)malloc(sizeof(float)*N);
b = (float*)malloc(sizeof(float)*N);
c = (float*)malloc(sizeof(float)*N);

for(int i=0;i<N;++i){
		a[i] = 1; b[i] = 2;

}
std::chrono::time_point<std::chrono::high_resolution_clock> start = std::chrono::high_resolution_clock::now();
vectorAddCPU(a,b,c,N);
std::chrono::time_point<std::chrono::high_resolution_clock> end = std::chrono::high_resolution_clock::now();
for(int i=0;i<N;++i){
		if(c[i] != 3.0f) {
				std::cout << "CPU nope\n";
				return -1;
		}
}
std::cout << "CPU OK -> " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " [ms]\n";
// Precisamos alocar memória na GPU
float *d_a,*d_b,*d_c;
hipMalloc((void**)&d_a,sizeof(float)*N);
hipMalloc((void**)&d_b,sizeof(float)*N);
hipMalloc((void**)&d_c,sizeof(float)*N);
hipMemcpy(d_a,a,sizeof(float)*N,hipMemcpyHostToDevice);
hipMemcpy(d_b,b,sizeof(float)*N,hipMemcpyHostToDevice);


int block_size = 256;
int grid_size = ((N+block_size) / block_size);
std::cout << "grid size = " << grid_size << std::endl;
//start = std::chrono::high_resolution_clock::now();
vectorAddGPU<<<grid_size,block_size>>>(d_a,d_b,d_c,N);
//end = std::chrono::high_resolution_clock::now();
hipDeviceSynchronize();
float * c_fromgpu = (float*)malloc(sizeof(float)*N);
hipMemcpy(c_fromgpu,d_c,sizeof(float)*N,hipMemcpyDeviceToHost);

std::cout << "GPU OK " << std::endl; 
std::cout << "Checking results .. " << std::endl;
 for(int i=0;i<N;++i){
 		if (c[i] != c_fromgpu[i])
 		{
 				std::cout << c[i] << "," << c_fromgpu[i] << std::endl;
 				std::cout << "Wrong results! :( " << i << std::endl;
 				return -1;
 		}
 }

std::cout << "All good!" << std::endl;
free(a);
free(b);
free(c);
free(c_fromgpu);

hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
}
