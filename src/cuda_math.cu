#include "hip/hip_runtime.h"
#include "cuda_math.h"
// For the CUDA runtime routines (prefixed with "cuda_")
// helper functions and utilities to work with CUDA

__global__ void vecAddKernel(const float *A, const float *B, float *C, int numElements) {

        int tid = threadIdx.x + blockIdx.x * blockDim.x;
        
        if(tid < numElements)
		    C[tid] = A[tid] + B[tid];
	
}

__global__ void singleThreadKernel(const float *A, const float *B, float *C,	int numElements) {

		for(int i=0;i<numElements;++i){
			C[i] = A[i] + B[i];
		}

	}


// C (m x k) = A (m x n)  * B (n x k)
__global__ void matMultKernel(const float* A, const float*B, float* C, int m, int n, int k){
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
	
    if( col < k && row < m) 
    {
		
        for(int i = 0; i < n; i++) 
        {
            sum += A[row * n + i] * B[i * k + col];
			
        }		
        C[row * k + col] = sum;	
    }
}



namespace gpu {


void vectorAdd(const float* A,const float *B, float* C, int N){

    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	

    
    // Mem Alloc
    float *d_a,*d_b,*d_c;
	hipError_t err;
	err = hipMalloc((void**)&d_a,sizeof(float)*N);
	if(err != hipSuccess){
		std::cout << "GPU allocation error " << std::endl;
			exit(EXIT_FAILURE);

	}

	err = hipMalloc((void**)&d_b,sizeof(float)*N);
	if(err != hipSuccess){
		std::cout << "GPU allocation error " << std::endl;
			exit(EXIT_FAILURE);

	}
	err = hipMalloc((void**)&d_c,sizeof(float)*N);
	if(err != hipSuccess){
		std::cout << "GPU allocation error " << std::endl;
			exit(EXIT_FAILURE);

	}


	printf("Total GPU memory: %ld Mbytes\n", sizeof(float)*N*3 / 1000000);


	err = hipMemcpy(d_a,A,sizeof(float)*N,hipMemcpyHostToDevice);
	if(err != hipSuccess){
		std::cout << "GPU memCpy error " << std::endl;
			exit(EXIT_FAILURE);

	}

	
	err = hipMemcpy(d_b,B,sizeof(float)*N,hipMemcpyHostToDevice);
	if(err != hipSuccess){
		std::cout << "GPU memCpy error " << std::endl;
			exit(EXIT_FAILURE);

	}

    
    int thread_per_block = 512; // MAX = 1024
    int grid_size = (N + thread_per_block - 1)/thread_per_block;
    
	
	// std::cout << "grid size = " << grid_size << std::endl;
	
    hipEventRecord(start, 0);

	
	singleThreadKernel<<<1,1>>>(d_a,d_b,d_c,N); // stupid

	// vecAddKernel<<<grid_size,thread_per_block>>>(d_a,d_b,d_c,N); // Launch N threads. MAX = 1024

	err = hipGetLastError();
	if(err != hipSuccess)
		std::cout << "Kernel launch error: " << hipGetErrorString(err) << std::endl;
	
	// vecAddKernel<<<grid_size,thread_per_block>>>(d_a,d_b,d_c,N);
	
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);

    // Results
    hipMemcpy(C,d_c,sizeof(float)*N,hipMemcpyDeviceToHost);


    float gpu_elapsed_time_ms;
    
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    
    std::cout << "GPU OK -> " << gpu_elapsed_time_ms << " [ms]\n";

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}

// C (m x k) = A (m x n)  * B (n x k)
void matrixMult(const float* A, const float*B, float*C, int m,int n,int k){
	float *d_a,*d_b,*d_c;
	// printf("%d %d %d \n",m,n,k);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipError_t err;
	hipMalloc(&d_a,sizeof(float)*m*n);
	hipMalloc(&d_b,sizeof(float)*n*k);
	hipMalloc(&d_c,sizeof(float)*m*k);

	printf("Total GPU memory: %ld Mbytes\n", (sizeof(float)*m*n + sizeof(float)*n*k + sizeof(float)*m*k) / 1000000);
	
	hipMemcpy(d_a,A,sizeof(float)*m*n,hipMemcpyHostToDevice);
	hipMemcpy(d_b,B,sizeof(float)*n*k,hipMemcpyHostToDevice);
	

	int thread_per_block = 2; // MAX = 1024
    unsigned int grid_rows = (m + thread_per_block - 1) / thread_per_block;
    unsigned int grid_cols = (k + thread_per_block - 1) / thread_per_block;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(thread_per_block, thread_per_block);

	printf("dimGrid: %d,%d\n\n",grid_cols,grid_rows);
	
	hipEventRecord(start, 0);
	matMultKernel<<<dimGrid, dimBlock>>>(d_a, d_b, d_c,m,n,k);

	err = hipGetLastError();
	if(err != hipSuccess)
		std::cout << "Kernel launch error: " << hipGetErrorString(err) << std::endl;
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float gpu_elapsed_time_ms;

	hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    
    std::cout << "GPU OK -> " << gpu_elapsed_time_ms << " [ms]\n";

	// Results
	hipMemcpy(C,d_c,sizeof(float)*m*k,hipMemcpyDeviceToHost);
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);


}




}