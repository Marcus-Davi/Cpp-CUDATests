#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <Eigen/Dense>

#define cudacall(call)                                                                                                          \
    do                                                                                                                          \
    {                                                                                                                           \
        hipError_t err = (call);                                                                                               \
        if(hipSuccess != err)                                                                                                  \
        {                                                                                                                       \
            fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
            hipDeviceReset();                                                                                                  \
            exit(EXIT_FAILURE);                                                                                                 \
        }                                                                                                                       \
    }                                                                                                                           \
    while (0)

#define cublascall(call)                                                                                        \
    do                                                                                                          \
    {                                                                                                           \
        hipblasStatus_t status = (call);                                                                         \
        if(HIPBLAS_STATUS_SUCCESS != status)                                                                     \
        {                                                                                                       \
            fprintf(stderr,"CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status);     \
            hipDeviceReset();                                                                                  \
            exit(EXIT_FAILURE);                                                                                 \
        }                                                                                                       \
                                                                                                                \
    }                                                                                                           \
    while(0)


int main(int argc,char ** argv){
    int Nmat;
    if (argc < 2)
        {
            printf("Usage: lin/solve [N elements]\n");
            exit(-1);
        }
    
        Nmat = atoi(argv[1]);
        std::cout << "N elements: " << Nmat << std::endl;
    	// Matrix Mult
	using Matrix = Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::ColMajor>;
	Matrix MA = Matrix::Random(Nmat, Nmat);
    Matrix b = Matrix::Random(Nmat,1);
    // std::cout << MA << std::endl << std::endl;
    // std::cout << b << std::endl << std::endl;
    // Matrix x = MA.colPivHouseholderQr().solve(b);

    Matrix MA_CPU = Matrix::Zero(Nmat, Nmat);  
    Matrix MA_GPU = Matrix::Zero(Nmat, Nmat);  
    
    // std::cout << x << std::endl;
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    float elapsed_time;
    float * src_d, *dst_d;

    // hipMalloc(&src_d,Nmat * Nmat * sizeof(float));
    // hipMemcpy(&src_d,MA.data(), Nmat*Nmat * sizeof(float),hipMemcpyHostToDevice);
    // hipMalloc(&dst_d,Nmat * Nmat * sizeof(float));

    hipEventRecord(start, 0);

    cudacall(hipMalloc<float>(&src_d,Nmat * Nmat * sizeof(float)));
    cudacall(hipMemcpy(src_d,MA.data(),Nmat * Nmat * sizeof(float),hipMemcpyHostToDevice));
    cudacall(hipMalloc<float>(&dst_d,Nmat * Nmat * sizeof(float)));



    hipblasHandle_t handle;
    cublascall(hipblasCreate(&handle));

    int batchSize = 1;

    int *P, *INFO;

    cudacall(hipMalloc<int>(&P,Nmat * batchSize * sizeof(int)));
    cudacall(hipMalloc<int>(&INFO,batchSize * sizeof(int)));

    int lda = Nmat;

    float *A[] = { src_d };
    float** A_d;
    cudacall(hipMalloc<float*>(&A_d,sizeof(A)));
    cudacall(hipMemcpy(A_d,A,sizeof(A),hipMemcpyHostToDevice));

    cublascall(hipblasSgetrfBatched(handle,Nmat,A_d,lda,P,INFO,batchSize));

    int INFOh = 0;
    cudacall(hipMemcpy(&INFOh,INFO,sizeof(int),hipMemcpyDeviceToHost));

    if(INFOh == Nmat)
    {
        fprintf(stderr, "Factorization Failed: Matrix is singular\n");
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }

    float* C[] = { dst_d };
    float** C_d;
    cudacall(hipMalloc<float*>(&C_d,sizeof(C)));
    cudacall(hipMemcpy(C_d,C,sizeof(C),hipMemcpyHostToDevice));

    cublascall(hipblasSgetriBatched(handle,Nmat,A_d,lda,P,C_d,lda,INFO,batchSize));

    cudacall(hipMemcpy(&INFOh,INFO,sizeof(int),hipMemcpyDeviceToHost));

    if(INFOh != 0)
    {
        fprintf(stderr, "Inversion Failed: Matrix is singular\n");
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }

    hipFree(P), hipFree(INFO), hipblasDestroy(handle);

    cudacall(hipMemcpy(MA_GPU.data(),dst_d,Nmat * Nmat * sizeof(float),hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

  
	hipEventElapsedTime(&elapsed_time, start, stop);
    std::cout << "GPU Inversion -> " << elapsed_time << " [ms]\n";

    hipEventRecord(start, 0);
    MA_CPU = MA.inverse();
    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	

	hipEventElapsedTime(&elapsed_time, start, stop);
    std::cout << "CPU Inversion -> " << elapsed_time << " [ms]\n";

    std::cout << "Checking Results..." << std::endl;
    for(int i = 0; i < MA.size() ;++i){
        if ( fabs( MA_GPU(i) - MA_CPU(i)) > 0.01 ) {
            std::cout << "Wrong results." << std::endl;
            exit(-1);
        }
    }

    std::cout << "OK!" << std::endl;

    // std::cout << MA_GPU << std::endl;
}